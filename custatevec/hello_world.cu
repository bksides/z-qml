#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}


int main(void) {
    printf("Hello from CPU!\n");
    cuda_hello<<<1,1>>>(); 
    return EXIT_SUCCESS;
}